#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#define IMAGE_DIM 2048
#define BOX_SIZE 1
#define ITERATIONS 100
#define NUMBER_OF_SAMPLES (((BOX_SIZE*2)+1)*((BOX_SIZE*2)+1))

void output_image_file(uchar4* image);
void input_image_file(char* filename, uchar4* image);
void checkCUDAError(const char *msg);

typedef enum { STARTING_CODE, EXERCISE_01, EXERCISE_02, EXERCISE_03, EXERCISE_04 } EXERCISE;

//The exercise mode can be set via pre-processor or by setting the `exercise` variable 
#ifdef EXERCISE_MODE
EXERCISE exercise = EXERCISE_MODE;
#elif
EXERCISE exercise = STARTING_CODE;
#endif

__global__ void image_blur_columns(uchar4 *image, uchar4 *image_output) {

	// map from threadIdx/BlockIdx to pixel row position
	int y = threadIdx.x + blockIdx.x * blockDim.x;

	//loop over columns
	for (int x = 0; x < IMAGE_DIM; x++){

		//calculate the input/output location
		int output_offset = x + y * IMAGE_DIM;
		uchar4 pixel;
		float4 average = make_float4(0, 0, 0, 0);

		for (int i = -BOX_SIZE; i <= BOX_SIZE; i++){
			for (int j = -BOX_SIZE; j <= BOX_SIZE; j++){
				int x_offset = x + i;
				int y_offset = y + j;
				//bounds check
				if ((x_offset < 0) || (x_offset >= IMAGE_DIM) || (y_offset < 0) || (y_offset >= IMAGE_DIM)){
					pixel = make_uchar4(0, 0, 0, 0);
				}
				else{
					//load pixel neighbour
					int offset = x_offset + y_offset * IMAGE_DIM;
					pixel = image[offset];
				}

				//sum values
				average.x += pixel.x;
				average.y += pixel.y;
				average.z += pixel.z;
			}
		}
		//calculate average
		average.x /= (float)NUMBER_OF_SAMPLES;
		average.y /= (float)NUMBER_OF_SAMPLES;
		average.z /= (float)NUMBER_OF_SAMPLES;

		image_output[output_offset].x = (unsigned char)average.x;
		image_output[output_offset].y = (unsigned char)average.y;
		image_output[output_offset].z = (unsigned char)average.z;
		image_output[output_offset].w = 255;
	}
}

/* Host code */

int main(void) {
	unsigned int image_size, i;
	uchar4 *d_image, *d_image_output, *d_image_temp;
	uchar4 *h_image;
	hipEvent_t start, stop;
	float3 ms; //[0]=normal,[1]=tex1d,[2]=tex2d

	image_size = IMAGE_DIM*IMAGE_DIM*sizeof(uchar4);

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on the GPU for the output image
	hipMalloc((void**)&d_image, image_size);
	hipMalloc((void**)&d_image_output, image_size);
	checkCUDAError("CUDA malloc");

	// allocate and load host image
	h_image = (uchar4*)malloc(image_size);
	input_image_file("input.ppm", h_image);

	switch (exercise){
	case(STARTING_CODE) : {
							  printf("Exercise Mode: Starting Code.\n");
                              // 1d by row
							  hipEventRecord(start, 0);
							  dim3    blocksPerGrid(IMAGE_DIM / 16, 1);
							  dim3    threadsPerBlock(16, 1);
							  // loop for number of iterations
							  for (i = 0; i < ITERATIONS; i++){
								  // copy image to device memory
								  hipMemcpy(d_image, h_image, image_size, hipMemcpyHostToDevice);
								  checkCUDAError("CUDA memcpy to device");

								  image_blur_columns << <blocksPerGrid, threadsPerBlock >> >(d_image, d_image_output);
								  checkCUDAError("kernel starting code implementation");

								  //copy results back to host
								  hipMemcpy(h_image, d_image_output, image_size, hipMemcpyDeviceToHost);
								  checkCUDAError("CUDA memcpy to host");

							  }
							  hipEventRecord(stop, 0);
							  hipEventSynchronize(stop);
							  hipEventElapsedTime(&ms.x, start, stop);
							  break;
	}
	case(EXERCISE_01) : {
                            printf("Exercise Mode: Exercise 01.\n");
							hipEventRecord(start, 0);
							dim3    blocksPerGrid(IMAGE_DIM / 16, 1);
							dim3    threadsPerBlock(16, 1);

							//TODO: Complete exercise 01

							hipEventRecord(stop, 0);
							hipEventSynchronize(stop);
							hipEventElapsedTime(&ms.x, start, stop);
							break;
	}
	case(EXERCISE_02) : {
							printf("Exercise Mode: Exercise 02.\n");
                            hipEventRecord(start, 0);
							dim3    blocksPerGrid(IMAGE_DIM / 16, 1);
							dim3    threadsPerBlock(16, 1);

							//TODO: Complete exercise 02

							hipEventRecord(stop, 0);
							hipEventSynchronize(stop);
							hipEventElapsedTime(&ms.x, start, stop);
							break;
	}
	case(EXERCISE_03) : {
							printf("Exercise Mode: Exercise 03.\n");
                            hipEventRecord(start, 0);
							dim3    blocksPerGrid(IMAGE_DIM / 16, IMAGE_DIM / 16);
							dim3    threadsPerBlock(16, 16);

							//TODO: Complete exercise 03

							hipEventRecord(stop, 0);
							hipEventSynchronize(stop);
							hipEventElapsedTime(&ms.x, start, stop);
							break;
	}
	}

	//output timings
	printf("Execution times:\n");
	printf("\tNormal version: %f\n", ms.x);

	// output image
	output_image_file(h_image);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_image);
	hipFree(d_image_output);
	free(h_image);

	return 0;
}

void output_image_file(uchar4* image)
{
	FILE *f; //output file handle

	//open the output file and write header info for PPM filetype
	f = fopen("output.ppm", "wb");
	if (f == NULL){
		fprintf(stderr, "Error opening 'output.ppm' output file\n");
		exit(1);
	}
	fprintf(f, "P6\n");
	fprintf(f, "# COM4521 Lab 05 Exercise02\n");
	fprintf(f, "%d %d\n%d\n", IMAGE_DIM, IMAGE_DIM, 255);
	for (int x = 0; x < IMAGE_DIM; x++){
		for (int y = 0; y < IMAGE_DIM; y++){
			int i = x + y*IMAGE_DIM;
			fwrite(&image[i], sizeof(unsigned char), 3, f); //only write rgb (ignoring a)
		}
	}

	fclose(f);
}

void input_image_file(char* filename, uchar4* image)
{
	FILE *f; //input file handle
	char temp[256];
	unsigned int x, y, s;

	//open the input file and write header info for PPM filetype
	f = fopen("input.ppm", "rb");
	if (f == NULL){
		fprintf(stderr, "Error opening 'input.ppm' input file\n");
		exit(1);
	}
	fscanf(f, "%s\n", &temp);
	fscanf(f, "%d %d\n", &x, &y);
	fscanf(f, "%d\n", &s);
	if ((x != y) && (x != IMAGE_DIM)){
		fprintf(stderr, "Error: Input image file has wrong fixed dimensions\n");
		exit(1);
	}

	for (int x = 0; x < IMAGE_DIM; x++){
		for (int y = 0; y < IMAGE_DIM; y++){
			int i = x + y*IMAGE_DIM;
			fread(&image[i], sizeof(unsigned char), 3, f); //only read rgb
			//image[i].w = 255;
		}
	}

	fclose(f);
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
